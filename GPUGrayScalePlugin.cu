#include "hip/hip_runtime.h"

#include "GPUGrayScalePlugin.h"

void GPUGrayScalePlugin::input(std::string myInputfile) {
  inputfile = myInputfile;
}

void GPUGrayScalePlugin::run() {

}

void GPUGrayScalePlugin::output(std::string outputfile) {
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  //args = gpuTKArg_read(argc, argv); /* parse the input arguments */

  hostInputImageData = readPPM(inputfile.c_str(), &imageWidth, &imageHeight);
  hostOutputImageData = (float*) malloc(3*imageWidth*imageHeight*sizeof(float));

  hipMalloc((void **)&deviceInputImageData,
             3*imageWidth * imageHeight * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             3*imageWidth * imageHeight * sizeof(float));
  hipMemcpy(deviceInputImageData, hostInputImageData,
             3*imageWidth * imageHeight * sizeof(float),
             hipMemcpyHostToDevice);

  ///////////////////////////////////////////////////////
  //@@ INSERT CODE HERE
  dim3 dimGrid(ceil((float)3*imageWidth / TILE_WIDTH),
               ceil((float)imageHeight / TILE_WIDTH));
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  rgb2gray<<<dimGrid, dimBlock>>>(deviceOutputImageData,
                                    deviceInputImageData, 3, imageWidth,
                                    imageHeight);

  ///////////////////////////////////////////////////////
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             3*imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  writePPM(outputfile.c_str(), imageWidth, imageHeight, hostOutputImageData);
  //gpuTKSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  free(hostInputImageData);
  free(hostOutputImageData);
}

PluginProxy<GPUGrayScalePlugin> GPUGrayScalePluginProxy = PluginProxy<GPUGrayScalePlugin>("GPUGrayScale", PluginManager::getInstance());

